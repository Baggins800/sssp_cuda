#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <stdio.h>
#define TPB 1024
#define INF 99999999

using namespace std;
struct Node {
  unsigned int start;
  unsigned int adj;
};
__global__ void intialize(unsigned int *c_dev,
                          bool *u_dev,
                          bool *f_dev,
                          unsigned int N) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N && tid > 0) {
    c_dev[tid] = INF;
    f_dev[tid] = false;
    u_dev[tid] = true;
  }
  if (tid == 0) {
    c_dev[tid] = 0;
    f_dev[tid] = true;
    u_dev[tid] = false;
  }
}
__global__ void relax_adj(unsigned int *c_dev, 
                          Node * v_dev, 
                          unsigned int *e_dev,
                          unsigned int *w_dev,
                          bool *u_dev,
                          unsigned int tid, 
                          unsigned int istart, 
                          unsigned int N) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if ( i < v_dev[tid].adj) {
    unsigned int succ = e_dev[i + istart];
    if (u_dev[succ]) {
      atomicMin(&c_dev[succ], c_dev[tid] + w_dev[i + istart]);
    }
  }
 
}
__global__ void relax_f(unsigned int *c_dev,
                        bool *u_dev,
                        bool *f_dev,
                        unsigned int *e_dev,
                        unsigned int *w_dev,
                        Node *v_dev,
                        unsigned int N) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    if (f_dev[tid]) {
      for (int i = v_dev[tid].start; 
           i < v_dev[tid].start + v_dev[tid].adj; 
           i++) {
        unsigned int succ = e_dev[i];
        if (u_dev[succ]) {
          atomicMin(&c_dev[succ], c_dev[tid] + w_dev[i]);
        }
      }
      //int blocks = v_dev[tid].adj / TPB + 1;
      //relax_adj<<<blocks, TPB>>>(c_dev, v_dev, e_dev, 
      //                           w_dev, u_dev, tid,
      //                           v_dev[tid].start, N);

    //hipDeviceSynchronize();
    }
  }
}
__global__ void update(unsigned int * c_dev,
                       bool *f_dev, bool *u_dev,
                       unsigned int mssp, unsigned int N) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    f_dev[tid] = false;
    if (c_dev[tid] == mssp) {
      u_dev[tid] = false;
      f_dev[tid] = true;
    }
  }
}
__global__ void minimum(unsigned int *c_dev,
                        bool *u_dev, unsigned int &mssp, unsigned int N) {
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    if (u_dev[tid] && (c_dev[tid] < mssp)) {
      atomicMin(&mssp, c_dev[tid]);
    }
  }
}
__device__ unsigned int mssp = 0;
__global__ void DA2CF(unsigned int *c_dev, 
           bool *u_dev, bool *f_dev, 
           unsigned int *e_dev, 
           unsigned int *w_dev,
           Node *v_dev,
           unsigned int N, vector<unsigned int> &P) {

  unsigned int extrablock = N % TPB > 0 ? 1 : 0;
  intialize<<<N / TPB + extrablock, TPB>>>(c_dev, u_dev, f_dev, N);
  hipDeviceSynchronize();
  while (mssp != INF) {
    mssp = INF;
    relax_f<<<N / TPB + extrablock, TPB>>>(c_dev, u_dev, f_dev, e_dev, w_dev, v_dev, N);
    hipDeviceSynchronize();
    minimum<<< N / TPB + extrablock, TPB >>>(c_dev, u_dev, mssp, N);   
    hipDeviceSynchronize();
    update<<< N / TPB + extrablock, TPB >>>(c_dev, f_dev, u_dev, mssp, N);
    hipDeviceSynchronize();
  }
  //for (int i = 0; i < N; i++) {
  // printf("%d ", c_dev[i]);
  //}
}

int main() {
  unsigned int N;
  unsigned int degree;
  unsigned int M;
  cin >> N;
  cin >> degree;
  cin >> M;
  vector<unsigned int> c_host(N);
  vector<Node> v_host(N);
  vector<unsigned int> e_host(M);
  vector<unsigned int> w_host(M);
  vector<unsigned int> P;
  unsigned int *c_dev, *w_dev, *e_dev;
  bool *f_dev, *u_dev;
  Node *v_dev;
  for (unsigned int i = 0; i < M; i++) {
    unsigned int ia, ib, w;
    cin >> ia >> ib >> w;
    e_host[i] = ib;
    w_host[i] = w;
  }
  for (unsigned int i = 0; i < N - degree; i++) {
     v_host[i].start = i * degree;
     v_host[i].adj = degree;
  }
  unsigned int dec = degree - 1;
  for (unsigned int z = N - degree; z < N; z++) {
    v_host[z].start = v_host[z - 1].start + dec + 1;
    v_host[z].adj = dec;
    dec--;
  }
  // allocate frontiers, unresolved and cost vectors on the GPU
  hipMalloc( (void**)&c_dev, N * sizeof(unsigned int) ); 
  hipMalloc( (void**)&f_dev, N * sizeof(bool) ); 
  hipMalloc( (void**)&u_dev, N * sizeof(bool) );
  hipMalloc( (void**)&v_dev, N * sizeof(Node) );
  hipMalloc( (void**)&e_dev, M * sizeof(unsigned int) );
  hipMalloc( (void**)&w_dev, M * sizeof(unsigned int) );

  // copy data to GPU memory
  hipMemcpy( v_dev, v_host.data(), N * sizeof(Node), hipMemcpyHostToDevice);
  hipMemcpy( e_dev, e_host.data(), M * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy( w_dev, w_host.data(), M * sizeof(unsigned int), hipMemcpyHostToDevice);
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventRecord(start,0);
  // execute dijkstra compound frontiers
  DA2CF<<<1,1>>>(c_dev, u_dev, f_dev, e_dev, w_dev, v_dev, N, P);
  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start,stop);
  cout << elapsedTime/1000.0f << " " << N << endl;

  // free allocated memory on the GPU
  hipFree(c_dev);
  hipFree(f_dev);
  hipFree(u_dev);
  hipFree(v_dev);
  hipFree(w_dev);
  hipFree(e_dev);
  return 0;
}
